#include "cusparseSpmvWrapper.hu"

using namespace thundercat;


CusparseSpmvWrapper::~CusparseSpmvWrapper() {
  hipFree(rowIndexDevPtr);
  hipFree(colIndexDevPtr);
  hipFree(valDevPtr);

  hipFree(x);
  hipFree(y);

  hipsparseDestroyMatDescr(descr);
  hipsparseDestroy(handle);

}

void CusparseSpmvWrapper::init() {
  handle = 0;
  descr = 0;

  hipsparseCreate(&handle);

  hipsparseStatus_t status = hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
}

void CusparseSpmvWrapper::preprocess(int nnz, int m, int n, int * rowPtr, int* colIdx, double* values) {

  M = m;
  N = n;
  NNZ = nnz;

  hipMalloc((void**)&rowIndexDevPtr, (N + 1) * sizeof(int));
  hipMalloc((void**)&colIndexDevPtr, NNZ * sizeof(int));
  hipMalloc((void**)&valDevPtr, NNZ * sizeof(double));

  hipMemcpy(rowIndexDevPtr, rowPtr,(size_t)((N + 1)*sizeof(rowPtr[0])),hipMemcpyHostToDevice);
  hipMemcpy(colIndexDevPtr, colIdx,(size_t)(NNZ*sizeof(colIdx[0])),hipMemcpyHostToDevice);
  hipMemcpy(valDevPtr, values,(size_t)(NNZ*sizeof(values[0])),hipMemcpyHostToDevice);

  hipMalloc((void**)&x, M * sizeof(double));
  hipMalloc((void**)&y, N * sizeof(double));


}

void CusparseSpmvWrapper::spmv(double * v, double * w) {
  double alpha = 1.0;
  double beta = 1.0;

  hipMemcpy(x, v,(size_t)(M*sizeof(v[0])),hipMemcpyHostToDevice);

  hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M, N, NNZ, &alpha,
                 descr, valDevPtr, rowIndexDevPtr, colIndexDevPtr, x, &beta, y);

  hipMemcpy(y, w,(size_t)(N*sizeof(y[0])),hipMemcpyDeviceToHost);

}