#include "cusparseSpmvWrapper.hu"

using namespace thundercat;

CusparseSpmvWrapper* thundercat::newCusparseSpmvWrapper() {
  return new CusparseSpmvWrapper();
};

void thundercat::deleteCusparseSpmvWrapper(CusparseSpmvWrapper* wrapper) {
  delete wrapper;
}

CusparseSpmvWrapper::~CusparseSpmvWrapper() {
  hipFree(rowIndexDevPtr);
  hipFree(colIndexDevPtr);
  hipFree(valDevPtr);

  hipFree(x);
  hipFree(y);

  hipsparseDestroyMatDescr(descr);
  hipsparseDestroy(handle);

}

void CusparseSpmvWrapper::init() {
  handle = 0;
  descr = 0;

  hipsparseCreate(&handle);

  hipsparseStatus_t status = hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
}

void CusparseSpmvWrapper::preprocess(int nnz, int m, int n, int * rowPtr, int* colIdx, double* values) {

  M = m;
  N = n;
  NNZ = nnz;

  hipError_t error = hipMalloc((void**)&rowIndexDevPtr, (N + 1) * sizeof(int));
  error = hipMalloc((void**)&colIndexDevPtr, NNZ * sizeof(int));
  error = hipMalloc((void**)&valDevPtr, NNZ * sizeof(double));

  error = hipMemcpy((void *)rowIndexDevPtr, (void*)rowPtr, (size_t) ((N + 1) * sizeof(int)), hipMemcpyHostToDevice);
  error = hipMemcpy((void*)colIndexDevPtr, (void*)colIdx, (size_t) (NNZ * sizeof(int)), hipMemcpyHostToDevice);
  error = hipMemcpy((void*) valDevPtr, (void*)values, (size_t) (NNZ * sizeof(double)), hipMemcpyHostToDevice);

  error = hipMalloc((void**)&x, M * sizeof(double));
  error = hipMalloc((void**)&y, N * sizeof(double));


}

void CusparseSpmvWrapper::spmv(double * v, double * w) {
  double alpha = 1.0;
  double beta = 1.0;

  hipMemcpy((void*) x, (void*) v,(size_t)(M*sizeof(double)),hipMemcpyHostToDevice);

  hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M, N, NNZ, &alpha,
                 descr, valDevPtr, rowIndexDevPtr, colIndexDevPtr, x, &beta, y);

  hipMemcpy((void*) w, (void*) y,(size_t)(N*sizeof(double)),hipMemcpyDeviceToHost);

}