#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include "cuspAdapter.hu"

using namespace thundercat;
void CuspAdapter::preprocess( int m, int n, int nnz, int * rowPtr, int * colIndx, double * values) {
  M = m;
  N = n;
  NNZ = nnz;

  hipMalloc(&devRowPtr, M * sizeof(int));
  hipMalloc(&devColIndx, NNZ * sizeof(int));
  hipMalloc(&devValues, NNZ * sizeof(double));
  hipMalloc(&devX, M * sizeof(double));
  hipMalloc(&devY, N * sizeof(double));

  hipMemcpy(devRowPtr,  rowPtr, M * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devColIndx, colIndx, NNZ * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devValues, values, NNZ * sizeof(double), hipMemcpyHostToDevice);

}

void CuspAdapter::spmv(double * v, double * w) {
  hipMemcpy(devX, v, M * sizeof(double), hipMemcpyHostToDevice);

    // *NOTE* raw pointers must be wrapped with thrust::device_ptr!
  thrust::device_ptr<int>   wrapped_device_Ap(devRowPtr);
  thrust::device_ptr<int>   wrapped_device_Aj(devColIndx);
  thrust::device_ptr<double> wrapped_device_Ax(devValues);
  thrust::device_ptr<double> wrapped_device_x(devX);
  thrust::device_ptr<double> wrapped_device_y(devY);

  // use array1d_view to wrap the individual arrays
  typedef typename cusp::array1d_view< thrust::device_ptr<int>   > DeviceIndexArrayView;
  typedef typename cusp::array1d_view< thrust::device_ptr<double> > DeviceValueArrayView;

  DeviceIndexArrayView row_offsets   (wrapped_device_Ap, wrapped_device_Ap + M);
  DeviceIndexArrayView column_indices(wrapped_device_Aj, wrapped_device_Aj + NNZ);
  DeviceValueArrayView values        (wrapped_device_Ax, wrapped_device_Ax + NNZ);
  DeviceValueArrayView x (wrapped_device_x, wrapped_device_x + M);
  DeviceValueArrayView y (wrapped_device_y, wrapped_device_y + N);


  typedef cusp::csr_matrix_view<DeviceIndexArrayView,
      DeviceIndexArrayView,
      DeviceValueArrayView> DeviceView;
  DeviceView A(M, N, NNZ, row_offsets, column_indices, values);


  cusp::multiply(A, x, y);
  hipMemcpy(w, devY, N * sizeof(double), hipMemcpyDeviceToHost);
}

CuspAdapter* thundercat::newCuspAdapter() {
  return new CuspAdapter();
}

void thundercat::deleteCuspAdapter(CuspAdapter* handle) {
  delete handle;
}
